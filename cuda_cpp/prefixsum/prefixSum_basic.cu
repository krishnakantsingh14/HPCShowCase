#include<iostream>
#include<vector>
#include <cassert>
#include<cstdlib>
#include<hip/hip_runtime.h>


__host__ void prefixsum_cpu(int* input_array, int* output_array, int shape) {
    int acc = input_array[0];
    output_array[0] = acc;
    for (int i = 1; i < shape; ++i) {
        acc += input_array[i];
        output_array[i] = acc;
    }

}

__global__ void  kogge_stone_scan_kernel(int* X, int* Y, int *blockSum, int shape) {

    extern __shared__ int XY[]; //The shape should be equal to blockdim 
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < shape) {
        XY[threadIdx.x] = X[tid];
    }
    else {
        XY[threadIdx.x] = 0; // Handle cases where tid >= shape
    }
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        int temp = 0;

        if (threadIdx.x >= stride && tid < shape)  {
            temp = XY[threadIdx.x - stride];
        }
        __syncthreads();
        XY[threadIdx.x] += temp;
        __syncthreads();

    }

    if (threadIdx.x == blockDim.x - 1) {
        blockSum[blockIdx.x] = XY[threadIdx.x];
    }

    if (tid < shape) {
        Y[tid] = XY[threadIdx.x];
    }
}

__global__ void reduced_blocks(int* Y, int *blockSum, int shape, int do_print =0) {
    //int B = blockIdx.x; 
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x>0 && tid < shape) {
        int sum = 0; 
        for (int i = 0; i < blockIdx.x; ++i) {
            sum += blockSum[i];
        }
        Y[tid] += sum;
        if (do_print ==1) {
            printf("%d, %d %d \n", sum, blockIdx.x, blockSum[blockIdx.x-1]);
        }
    }

}


__host__ void check_array(int* first_array, int* second_array, int size) {

    for (int i = 0; i < size; ++i) {
        if (first_array[i] != second_array[i]) {
            std::cerr << "Arrays differ at index " << i << ": "
                << "arr1[" << i << "] = " << first_array[i] << ", "
                << "arr2[" << i << "] = " << second_array[i] << std::endl;
        }
    }
    std::cout << "SUCCESS" << std::endl;
}



__host__ void initialize_array(int* inputarray, int size) {

    for (int i = 0; i < size; ++i) {
        inputarray[i] = 1;// rand() % 100; //random numbers from 0 to 100;
    }
}


int main() {
    const int SHAPE = 1 << 8;  // 256
    const size_t bytesOfArray = SHAPE * sizeof(int);

    // Allocate and initialize arrays
    int* inp_array = nullptr;
    int* out_cpu_array = new int[SHAPE];
    int* out_gpu_array = nullptr;

    hipMallocManaged(&inp_array, bytesOfArray);
    hipMallocManaged(&out_gpu_array, bytesOfArray);

    initialize_array(inp_array, SHAPE);

    // Initialize output arrays
    std::fill(out_cpu_array, out_cpu_array + SHAPE, 0);
    std::fill(out_gpu_array, out_gpu_array + SHAPE, 0);

    // CPU Prefix Sum
    prefixsum_cpu(inp_array, out_cpu_array, SHAPE);
    std::cout << "CPU computation done." << std::endl;

    // GPU Prefix Sum using Kogge-Stone scan
    const int blockSize = 128;
    const int numBlocks = (SHAPE + blockSize - 1) / blockSize;
    const size_t sharedMemSize = blockSize * sizeof(int);

    int* block_sums = nullptr;
    hipMalloc(&block_sums, numBlocks * sizeof(int));

    // Launching the kernels
    kogge_stone_scan_kernel<<<numBlocks, blockSize, sharedMemSize>>>(inp_array, out_gpu_array, block_sums, SHAPE);
    reduced_blocks<<<numBlocks, blockSize>>>(out_gpu_array, block_sums, SHAPE);

    // Synchronize to wait for the GPU to finish
    hipDeviceSynchronize();
    std::cout << "GPU computation done." << std::endl;

    // Check the results
    check_array(out_cpu_array, out_gpu_array, SHAPE);

    // Clean up resources
    hipFree(inp_array);
    hipFree(out_gpu_array);
    hipFree(block_sums);
    delete[] out_cpu_array;

    return 0;
}